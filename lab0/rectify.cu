#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "lodepng.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void rectification(unsigned char* image, unsigned char* new_image, unsigned int size)
{
    //change this so it doesnt look too conspicuous
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < size) {
        
        if (image[index] < 127) {
            new_image[index] = 127;
        }
        else {
            new_image[index] = image[index];
        }
    }

}


int main(int argc, char* argv[])
{

    // get arguments from command line
    char* input_filename = "Test Images\\Test_1.png"; //argv[1];
    char* output_filename = "Output Images\\Test_1_output.png"; //argv[2];
    int threadNum = 1000; //atoi(argv[3]);
    
    //getting image and its size
    unsigned error;
    unsigned char* image, * new_image_rec;
    unsigned width, height;
    
    error = lodepng_decode32_file(&image, &width, &height, input_filename);
    if (error) printf("error %u: %s\n", error, lodepng_error_text(error));
    unsigned int size = width * height * 4 * sizeof(unsigned char);
    new_image_rec = (unsigned char*)malloc(size);

    //defining device vars
    unsigned char* image_cuda, * new_image_rec_cuda;
    hipMalloc((void**) &image_cuda, size);
    hipMalloc((void**) &new_image_rec_cuda, size);
    hipMemcpy(image_cuda, image, size, hipMemcpyHostToDevice);

    //start timer
    float memsettime;
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start, 0);

    //rectify
    rectification << < (size + threadNum - 1) / threadNum, threadNum >> > (image_cuda, new_image_rec_cuda, size);
    hipDeviceSynchronize();
 
    //stop timer
    hipEventRecord(stop, 0); hipEventSynchronize(stop);
    hipEventElapsedTime(&memsettime, start, stop);
    printf("Rectify: thread count is %d, ran in %f milliseconds\n", threadNum, memsettime);
    hipEventDestroy(start); hipEventDestroy(stop);

    //free cuda memory
    hipMemcpy(new_image_rec, new_image_rec_cuda, size, hipMemcpyDeviceToHost);
    hipFree(image_cuda);
    hipFree(new_image_rec_cuda);

    //save png image
    lodepng_encode32_file(output_filename, new_image_rec, width, height);
    
    //free memory
    free(image);
    free(new_image_rec);

    return 0;
}


