#include "hip/hip_runtime.h"
/*
* ECSE420 LAB1: Group 15, Sabina Sasu & Erica De Petrillo
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <Math.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

__global__ void logic_gate_explicit(char* data, int file_length, char* outputData, int threadNum)
{

	for (int i = threadIdx.x + (blockIdx.x * blockDim.x); i < file_length; i += threadNum) {
		//printf("index: %d\n", i);
		int gateA = data[i * 6] - '0';
		int gateB = data[i * 6 + 2] - '0';
		int gateNum = data[i * 6 + 4] - '0';
		//printf("%d, %d, %d\n", gateA, gateB, gateNum);
		int output = 0;
		switch (gateNum) {
		case AND: output = gateA & gateB;  break;
		case OR: output = gateA | gateB; break;
		case NAND: output = !(gateA & gateB); break;
		case NOR: output = !(gateA | gateB); break;
		case XOR: output = ((!gateA & gateB) | (gateA & !gateB)); break;
		case XNOR: output = !((!gateA & gateB) | (gateA & !gateB)); break;
		}
		//printf("%d\n", output);
		outputData[i * 2] = output + '0';
		outputData[i * 2 + 1] = '\n';
	}
}


int process_explicit(int argc, char* argv[]) {

	if (argc != 4)
		return 0;

	// get arguments from command line
	char* input_filename = argv[1];
	int file_length = atoi(argv[2]);
	char* output_filename = argv[3];

	FILE* input_file;
	FILE* output_file;

	if ((input_file = fopen(input_filename, "r")) == NULL) {
		printf("Error! opening file");
		// Program exits if file pointer returns NULL.
		exit(1);
	}

	if ((output_file = fopen(output_filename, "w")) == NULL) {
		printf("Error! opening file");
		// Program exits if file pointer returns NULL.
		exit(1);
	}

	int num_blocks = 1;
	int num_threads_per_block = file_length;

	if (file_length > 1024) {
		num_blocks = ((file_length - 1) / 1024) + 1; //1024 is the max number of threads in 1 block
		num_threads_per_block = ceil(file_length / num_blocks);
	}

	//create timer for data migration
	float memsettime2;
	hipEvent_t start2, stop2;
	hipEventCreate(&start2); hipEventCreate(&stop2);
	hipEventRecord(start2, 0);

	char* data;
	char* output;
	char* d_data;
	char* d_output;
	data = (char*)malloc(file_length * 6);
	output = (char*)malloc(file_length * 2);
	hipMalloc(&d_data, file_length * 6);
	hipMalloc(&d_output, file_length * 2);
	fread(data, 1, file_length * 6, input_file);
	hipMemcpy(d_data, data, file_length * 6, hipMemcpyHostToDevice);
	hipMemcpy(d_output, output, file_length * 2, hipMemcpyHostToDevice);

	hipEventRecord(stop2, 0); hipEventSynchronize(stop2);
	hipEventElapsedTime(&memsettime2, start2, stop2);
	printf("Parallel Explicit data migration: file_length is %d, ran in %f milliseconds\n", file_length, memsettime2);
	hipEventDestroy(start2); hipEventDestroy(stop2);
	
	//start timer for execution runtime 
	float memsettime;
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// run
	logic_gate_explicit << < num_blocks, num_threads_per_block >> > (d_data, file_length, d_output, num_threads_per_block);

	//stop timer
	hipEventRecord(stop, 0); hipEventSynchronize(stop);
	hipEventElapsedTime(&memsettime, start, stop);
	printf("Parallel Explicit: file_length is %d, ran in %f milliseconds\n", file_length, memsettime);
	hipEventDestroy(start); hipEventDestroy(stop);

	//free cuda memory
	hipMemcpy(output, d_output, file_length * 2, hipMemcpyDeviceToHost);
	hipFree(d_data);
	hipFree(d_output);

	//write output to file
	fwrite(output, 1, file_length * 2, output_file);

	fclose(input_file);
	fclose(output_file);
	free(data);
	free(output);

	return 0;
}

int main(int argc, char* argv[]) { return process_explicit(argc, argv); }
