#include "hip/hip_runtime.h"
/*
* ECSE420 LAB0: Group 15, Sabina Sasu & Erica De Petrillo
*/

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

__global__ void logic_gate(char* data, int file_length, char * outputData, int threadNum)
{

	for (int i = threadIdx.x + blockIdx.x; i < file_length; i += threadNum) {
		//printf("index: %d\n", i);
		int gateA = data[i*6] - '0';
		int gateB = data[i*6+2] - '0';
		int gateNum = data[i*6+4] - '0';
		//printf("%d, %d, %d\n", gateA, gateB, gateNum);
		int output = 0;
		switch (gateNum) {
			case AND: output = gateA & gateB;  break;
			case OR: output = gateA | gateB; break;
			case NAND: output = !(gateA & gateB); break;
			case NOR: output = !(gateA | gateB); break;
			case XOR: output = ((!gateA & gateB) | (gateA & !gateB)); break;
			case XNOR: output = !((!gateA & gateB) | (gateA & !gateB)); break;
		}
		//printf("%d\n", output);
		outputData[i*2] = output +'0';
		outputData[i * 2 +1] = '\n';
	}
}


int process_unified(int argc, char* argv[]) {

	//if (argc != 4)
	//	return 0;

	// get arguments from command line
	char* input_filename = "test_data\\input_100000.txt";//argv[1];
	int file_length = 10000;//atoi(argv[2]);
	char* output_filename = "output_data\\output_100000.txt";//argv[3];

	FILE* input_file;
	FILE* output_file;

	if ((input_file = fopen(input_filename, "r")) == NULL) {
		printf("Error! opening file");
		// Program exits if file pointer returns NULL.
		exit(1);
	}

	if ((output_file = fopen(output_filename, "w")) == NULL) {
		printf("Error! opening file");
		// Program exits if file pointer returns NULL.
		exit(1);
	}

	int threadNum = file_length;
	int blockNum = 1;
	if (file_length > 1024) {
		threadNum = 1024;
		blockNum = file_length / 1024;
		printf("%d\n", threadNum);
		printf("%d\n", blockNum);
	}

	char* data;
	char* output;
	hipMallocManaged(&data, file_length*6);
	hipMallocManaged(&output, file_length*2);
	fread(data, 1, file_length*6, input_file);
	//printf("%s\n", data);
	//start timer
	float memsettime;
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// pool
	logic_gate << < blockNum, threadNum >> > (data, file_length, output, threadNum);
	hipDeviceSynchronize();

	//stop timer
	hipEventRecord(stop, 0); hipEventSynchronize(stop);
	hipEventElapsedTime(&memsettime, start, stop);
	printf("Pool: thread count is %d, ran in %f milliseconds\n", threadNum, memsettime);
	hipEventDestroy(start); hipEventDestroy(stop);

	//free cuda memory
	//printf("output:\n %s\n", output);
	
	fwrite(output, 1, file_length*2, output_file);
	
	fclose(input_file);
	fclose(output_file);
	hipFree(data);
	hipFree(output);
	return 0;
}

//int main(int argc, char* argv[]) { return process_unified(argc, argv); }
