#include "hip/hip_runtime.h"
/*
* ECSE420 LAB3: Group 15, Sabina Sasu & Erica De Petrillo
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

int read_input_one_two_four2(int** input1, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}
	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	int temp1;
	while (fscanf(fp, "%d", &temp1) == 1) {
		(*input1)[counter] = temp1;

		counter++;
	}
	fclose(fp);
	return len;
}

int read_input_three2(int** input1, int** input2, int** input3, int** input4, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}
	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	*input2 = (int*)malloc(len * sizeof(int));
	*input3 = (int*)malloc(len * sizeof(int));
	*input4 = (int*)malloc(len * sizeof(int));
	int temp1;
	int temp2;
	int temp3;
	int temp4;
	while (fscanf(fp, "%d,%d,%d,%d", &temp1, &temp2, &temp3, &temp4) == 4) {
		(*input1)[counter] = temp1;
		(*input2)[counter] = temp2;
		(*input3)[counter] = temp3;
		(*input4)[counter] = temp4;
		counter++;
	}
	fclose(fp);
	return len;
}

__device__ int device_count = 0;
__global__ void block_queuing_kernel(int block_queue_capacity, int threadNum, int numCurrLevelNodes, int* numNextLevelNodes_h,
	int* currLevelNodes_h, int* nodePtrs_h, int* nodeNeighbors_h, int* nodeVisited_h,
	int* nodeGate_h, int* nodeInput_h, int* nodeOutput_h, int* nextLevelNodes_h) {

	extern __shared__ int shared_mem_queue[];
	int i = threadIdx.x + (blockIdx.x * blockDim.x);	
	int stride = blockDim.x * gridDim.x;
	__shared__ int counter; //counter = 0 here
	//printf("counter: %d\n", counter);
	//printf("%d\n", sizeof(shared_mem_queue) / sizeof(int));

	// Loop over all nodes in the current level
	for (int idx = i; idx < numCurrLevelNodes; idx += stride) {
		int node = currLevelNodes_h[idx];
		// Loop over all neighbors of the node
		for (int nbrIdx = nodePtrs_h[node]; nbrIdx < nodePtrs_h[node + 1]; nbrIdx++) {
			int neighbor = nodeNeighbors_h[nbrIdx];
			// If the neighbor hasn't been visited yet
			if (!nodeVisited_h[neighbor]) {
				// Mark it and add it to the queue
				nodeVisited_h[neighbor] = 1;
				
				//solve gate
				int result = 0;
				int output = nodeOutput_h[node];
				int input = nodeInput_h[neighbor];
				switch (nodeGate_h[neighbor]) {
					case AND: result = output & input;  break;
					case OR: result = output | input; break;
					case NAND: result = !(output & input); break;
					case NOR: result = !(output | input); break;
					case XOR: result = ((!output & input) | (output & !input)); break;
					case XNOR: result = !((!output & input) | (output & !input)); break;
				}

				nodeOutput_h[neighbor] = result;
				//atomicAdd(&counter, 1); //we are going to add an entry to the shared mem queue
				//printf("right after add\n");
				//printf("counter: %d\n", counter);
				if (counter >= block_queue_capacity) { //queue full
					//printf("inside if before\n");
					nextLevelNodes_h[atomicAdd(&device_count, 1)] = neighbor;
					*numNextLevelNodes_h = device_count;
					//printf("inside if\n");
				}
				else {
					shared_mem_queue[atomicAdd(&counter, 1)] = neighbor; //adding neighbor to shared mem queue
					__syncthreads();
					//nextLevelNodes_h[*numNextLevelNodes_h] = neighbor;
					
				}
				//printf("before sync\n");
				__syncthreads();
				//printf("after sync\n");
				//++(*numNextLevelNodes_h);
				
				//printf("after ++\n");
				//printf("here: %d\n", *numNextLevelNodes_h);
			}
		}
	}
	//printf("hello");
	//for (int i = 0; i < block_queue_capacity; i++) {
	//	printf("%d\n", shared_mem_queue[i]);
	//}
	//allocate space for block queue to go into global queue
	//store block queue in global queue
	for (int i = 0; i < block_queue_capacity; i++) {
		printf("inside else %d\n", shared_mem_queue[i]);
		nextLevelNodes_h[atomicAdd(&device_count, 1)] = shared_mem_queue[i];
		*numNextLevelNodes_h = device_count;
	}
}



int process_block(int argc, char* argv[]) {

	//if (argc != 7)
	//	return 0;
	// get arguments from command line
	char* input_filename1 = "input1.raw";//argv[1];
	char* input_filename2 = "input2.raw";//argv[2];
	char* input_filename3 = "input3.raw";//argv[3];
	char* input_filename4 = "input4.raw";//argv[4];
	char* output_node_filename = "output/output_node.raw";//argv[5];
	char* output_next_node_filename = "output/output_next_node.raw";//argv[6];

	int mode = 1; //can be mode 1 or 2
	//number of threads
	int num_of_threads = 0;
	int num_of_blocks = 0;
	int block_queue_capacity = 0;

	if (mode == 1) { //32 threads per block, 25 blocks, 32 queue capacity
		num_of_threads = 1;
		num_of_blocks = 10;
		block_queue_capacity = 32;
	}
	else if (mode == 2) { //64 threads per block, 35 blocks, 64 queue capacity
		num_of_threads = 64;
		num_of_blocks = 35;
		block_queue_capacity = 64;
	}
	
	//Variables
	int numNodePtrs;
	int numNodes;
	int* nodePtrs_h;
	int* nodeNeighbors_h;
	int* nodeVisited_h;
	int numTotalNeighbors_h;
	int* currLevelNodes_h;
	int numCurrLevelNodes;
	int* numNextLevelNodes_h;
	int* nodeGate_h;
	int* nodeInput_h;
	int* nodeOutput_h;

	numNodePtrs = read_input_one_two_four2(&nodePtrs_h, input_filename1);
	numTotalNeighbors_h = read_input_one_two_four2(&nodeNeighbors_h, input_filename2);
	numNodes = read_input_three2(&nodeVisited_h, &nodeGate_h, &nodeInput_h, &nodeOutput_h, input_filename3);
	numCurrLevelNodes = read_input_one_two_four2(&currLevelNodes_h, input_filename4);

	//output
	int* nextLevelNodes_h = (int*)malloc(numNodePtrs * sizeof(int));

	//initialize cuda vars
	int* currLevelNodes_c, int* nodePtrs_c, int* nodeNeighbors_c, int* nodeVisited_c,
		int* nodeGate_c, int* nodeInput_c, int* nodeOutput_c, int* nextLevelNodes_c;
	hipMalloc(&currLevelNodes_c, numCurrLevelNodes * sizeof(int));
	hipMalloc(&nodePtrs_c, numNodePtrs * sizeof(int));
	hipMalloc(&nodeNeighbors_c, numTotalNeighbors_h * sizeof(int));
	hipMalloc(&nodeVisited_c, numNodes * sizeof(int));
	hipMalloc(&nodeGate_c, numNodes * sizeof(int));
	hipMalloc(&nodeInput_c, numNodes * sizeof(int));
	hipMalloc(&nodeOutput_c, numNodes * sizeof(int));
	hipMalloc(&nextLevelNodes_c, numNodes * sizeof(int));

	hipMemcpy(currLevelNodes_c, currLevelNodes_h, numCurrLevelNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nodePtrs_c, nodePtrs_h, numNodePtrs * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nodeNeighbors_c, nodeNeighbors_h, numTotalNeighbors_h * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nodeVisited_c, nodeVisited_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nodeGate_c, nodeGate_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nodeInput_c, nodeInput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nodeOutput_c, nodeOutput_h, numNodes * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(nextLevelNodes_c, nextLevelNodes_h, numNodes * sizeof(int), hipMemcpyHostToDevice);

	hipMallocManaged(&numNextLevelNodes_h, sizeof(int));
	*numNextLevelNodes_h = 0;

	//start timer for execution runtime 
	float memsettime;
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start, 0);

	block_queuing_kernel << < num_of_blocks, num_of_threads, (block_queue_capacity * sizeof(int)) >> > (block_queue_capacity, num_of_threads, numCurrLevelNodes, numNextLevelNodes_h,
			currLevelNodes_c, nodePtrs_c, nodeNeighbors_c, nodeVisited_c, nodeGate_c, nodeInput_c, nodeOutput_c, nextLevelNodes_c);

	hipDeviceSynchronize();

	//stop timer
	hipEventRecord(stop, 0); hipEventSynchronize(stop);
	hipEventElapsedTime(&memsettime, start, stop);
	printf("Block queueing: ran in %f milliseconds\n", memsettime);
	hipEventDestroy(start); hipEventDestroy(stop);


	//free cuda memory
	hipMemcpy(nodeOutput_h, nodeOutput_c, (numNodePtrs - 1) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(nextLevelNodes_h, nextLevelNodes_c, *numNextLevelNodes_h * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(currLevelNodes_c);
	hipFree(nodePtrs_c);
	hipFree(nodeNeighbors_c);
	hipFree(nodeVisited_c);
	hipFree(nodeGate_c);
	hipFree(nodeInput_c);
	hipFree(nodeOutput_c);
	hipFree(nextLevelNodes_c);


	//output results to respective files
	FILE* output_file_node;
	FILE* output_file_next;
	if ((output_file_node = fopen(output_node_filename, "w")) == NULL) {
		printf("Error! opening file");
		exit(1);
	}
	if ((output_file_next = fopen(output_next_node_filename, "w")) == NULL) {
		printf("Error! opening file");
		exit(1);
	}

	fprintf(output_file_node, "%d\n", numNodePtrs - 1);
	for (int loop = 0; loop < numNodePtrs - 1; loop++)
		fprintf(output_file_node, "%d\n", nodeOutput_h[loop]);
	fclose(output_file_node);

	fprintf(output_file_next, "%d\n", *numNextLevelNodes_h);
	for (int loop = 0; loop < *numNextLevelNodes_h; loop++)
		fprintf(output_file_next, "%d\n", nextLevelNodes_h[loop]);
	fclose(output_file_next);

	return 0;
}

int main(int argc, char* argv[]) { return process_block(argc, argv); }
