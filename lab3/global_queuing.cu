#include "hip/hip_runtime.h"
/*
* ECSE420 LAB3: Group 15, Sabina Sasu & Erica De Petrillo
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

int read_input_one_two_four(int** input1, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}
	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	int temp1;
	while (fscanf(fp, "%d", &temp1) == 1) {
		(*input1)[counter] = temp1;

		counter++;
	}
	fclose(fp);
	return len;
}

int read_input_three(int** input1, int** input2, int** input3, int** input4, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}
	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	*input2 = (int*)malloc(len * sizeof(int));
	*input3 = (int*)malloc(len * sizeof(int));
	*input4 = (int*)malloc(len * sizeof(int));
	int temp1;
	int temp2;
	int temp3;
	int temp4;
	while (fscanf(fp, "%d,%d,%d,%d", &temp1, &temp2, &temp3, &temp4) == 4) {
		(*input1)[counter] = temp1;
		(*input2)[counter] = temp2;
		(*input3)[counter] = temp3;
		(*input4)[counter] = temp4;
		counter++;
	}
	fclose(fp);
	return len;
}


__global__ void global_queuing_kernel(...){
	int i = threadIdx.x + (blockIdx.x * blockDim.x);

	//im guessing this is the same as sequential but we loop over a particular interval of nodes based on thread number?

	// Loop over all nodes in the current level
	for (int idx = i; idx < numCurrLevelNodes; idx++) {
		int node = currLevelNodes_h[idx];
		// Loop over all neighbors of the node
		for (int nbrIdx = nodePtrs_h[node]; nbrIdx < nodePtrs_h[node + 1]; nbrIdx++) {
			int neighbor = nodeNeighbors_h[nbrIdx];
			// If the neighbor hasn't been visited yet
			if (!nodeVisited_h[neighbor]) {
				// Mark it and add it to the queue
				nodeVisited_h[neighbor] = 1;
				nodeOutput_h[neighbor] = gate_solver(nodeGate_h[neighbor], nodeOutput_h[node], nodeInput_h[neighbor]);
				nextLevelNodes_h[numNextLevelNodes_h] = neighbor;
				++(numNextLevelNodes_h);
			}
		}
	}
}



int gate_solver(int gate, int output, int input) {
	int result = 0;
	switch (gate) {
	case AND: result = output & input;  break;
	case OR: result = output | input; break;
	case NAND: result = !(output & input); break;
	case NOR: result = !(output | input); break;
	case XOR: result = ((!output & input) | (output & !input)); break;
	case XNOR: result = !((!output & input) | (output & !input)); break;
	}
	return result;
}

int process_block(int argc, char* argv[]) {

	//if (argc != 7)
	//	return 0;
	// get arguments from command line
	char* input_filename1 = "input1.raw";//argv[1];
	char* input_filename2 = "input2.raw";//argv[2];
	char* input_filename3 = "input3.raw";//argv[3];
	char* input_filename4 = "input4.raw";//argv[4];
	char* output_node_filename = "output/output_node.raw";//argv[5];
	char* output_next_node_filename = "output/output_next_node.raw";//argv[6];

	int mode = 1;
	//number of threads
	int num_of_threads = 0;
	int num_of_blocks = 0;

	if (mode == 1) { //16 blocks, 16 elements by thread, 32 rows per block, 32 threads per row
		num_of_threads = 10;
		num_of_blocks = 32;
	}
	else if (mode == 2) { //512 blocks, 1 element by thread, 1 row per block, 512 threads per row
		num_of_threads = 25;
		num_of_blocks = 32;
	}
	else if (mode == 3) { //64 blocks, 4 elements by thread, 8 rows per block, 128 threads per row
		num_of_threads = 25;
		num_of_blocks = 64;
	}
	else if (mode == 4) { //64 blocks, 4 elements by thread, 8 rows per block, 128 threads per row
		num_of_threads = 35;
		num_of_blocks = 64;
	}
	else if (mode == 5) { //64 blocks, 4 elements by thread, 8 rows per block, 128 threads per row
		num_of_threads = 35;
		num_of_blocks = 128;
	}

	//Code provided:
	//Variables
	int numNodePtrs;
	int numNodes;
	int* nodePtrs_h;
	int* nodeNeighbors_h;
	int* nodeVisited_h;
	int numTotalNeighbors_h;
	int* currLevelNodes_h;
	int numCurrLevelNodes;
	int numNextLevelNodes_h = 0;
	int* nodeGate_h;
	int* nodeInput_h;
	int* nodeOutput_h;

	numNodePtrs = read_input_one_two_four(&nodePtrs_h, input_filename1);
	numTotalNeighbors_h = read_input_one_two_four(&nodeNeighbors_h, input_filename2);
	numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h, &nodeOutput_h, input_filename3);
	numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, input_filename4);

	//output
	int* nextLevelNodes_h = (int*)malloc(numNodePtrs * sizeof(int));


	//start timer for execution runtime 
	float memsettime;
	hipEvent_t start, stop;
	hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord(start, 0);

	global_queuing_kernel << < num_of_blocks, num_of_threads >> > (...);

	//stop timer
	hipEventRecord(stop, 0); hipEventSynchronize(stop);
	hipEventElapsedTime(&memsettime, start, stop);
	printf("Global queueing: ran in %f milliseconds\n", memsettime);
	hipEventDestroy(start); hipEventDestroy(stop);

	//output results to respective files

	FILE* output_file_node;
	FILE* output_file_next;
	if ((output_file_node = fopen(output_node_filename, "w")) == NULL) {
		printf("Error! opening file");
		exit(1);
	}
	if ((output_file_next = fopen(output_next_node_filename, "w")) == NULL) {
		printf("Error! opening file");
		exit(1);
	}
	//first line is the length
	fprintf(output_file_node, "%d\n", numNodePtrs - 1);
	for (int loop = 0; loop < numNodePtrs - 1; loop++)
		fprintf(output_file_node, "%d\n", nodeOutput_h[loop]);
	fclose(output_file_node);

	fprintf(output_file_node, "%d\n", numNextLevelNodes_h);
	for (int loop = 0; loop < numNextLevelNodes_h; loop++)
		fprintf(output_file_next, "%d\n", nextLevelNodes_h[loop]);


	fclose(output_file_next);

	return 0;
}

int main(int argc, char* argv[]) { return process_block(argc, argv); }
