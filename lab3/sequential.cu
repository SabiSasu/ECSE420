/*
* ECSE420 LAB3: Group 15, Sabina Sasu & Erica De Petrillo
*/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#define AND 0
#define OR 1
#define NAND 2
#define NOR 3
#define XOR 4
#define XNOR 5

int read_input_one_two_four(int** input1, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}
	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	int temp1;
	while (fscanf(fp, "%d", &temp1) == 1) {
		(*input1)[counter] = temp1;

		counter++;
	}
	fclose(fp);
	return len;
}

int read_input_three(int** input1, int** input2, int** input3, int** input4, char* filepath) {
	FILE* fp = fopen(filepath, "r");
	if (fp == NULL) {
		fprintf(stderr, "Couldn't open file for reading\n");
		exit(1);
	}
	int counter = 0;
	int len;
	int length = fscanf(fp, "%d", &len);
	*input1 = (int*)malloc(len * sizeof(int));
	*input2 = (int*)malloc(len * sizeof(int));
	*input3 = (int*)malloc(len * sizeof(int));
	*input4 = (int*)malloc(len * sizeof(int));
	int temp1;
	int temp2;
	int temp3;
	int temp4;
	while (fscanf(fp, "%d,%d,%d,%d", &temp1, &temp2, &temp3, &temp4) == 4) {
		(*input1)[counter] = temp1;
		(*input2)[counter] = temp2;
		(*input3)[counter] = temp3;
		(*input4)[counter] = temp4;
		counter++;
	}
	fclose(fp);
	return len;
}

int gate_solver(int gate, int output, int input) {
	int result = 0;
	switch (gate) {
		case AND: result = output & input;  break;
		case OR: result = output | input; break;
		case NAND: result = !(output & input); break;
		case NOR: result = !(output | input); break;
		case XOR: result = ((!output & input) | (output & !input)); break;
		case XNOR: result = !((!output & input) | (output & !input)); break;
	}
	return result;
}

int process_sequential(int argc, char* argv[]) {

	char* input_filename1 = "input1.raw";//argv[1];
	char* input_filename2 = "input2.raw";//argv[2];
	char* input_filename3 = "input3.raw";//argv[3];
	char* input_filename4 = "input4.raw";//argv[4];
	char* output_node_filename = "output/output_node.raw";//argv[5];
	char* output_next_node_filename = "output/output_next_node.raw";//argv[6];

	//if we have the arguments from cmd, take them instead
	if (argc == 7) {
		char* input_filename1 = argv[1];
		char* input_filename2 = argv[2];
		char* input_filename3 = argv[3];
		char* input_filename4 = argv[4];
		char* output_node_filename = argv[5];
		char* output_next_node_filename = argv[6];
	}
																	
	//Code provided:
	//Variables
	int numNodePtrs;
	int numNodes;
	int* nodePtrs_h;
	int* nodeNeighbors_h;
	int* nodeVisited_h;
	int numTotalNeighbors_h;
	int* currLevelNodes_h;
	int numCurrLevelNodes;
	int numNextLevelNodes_h = 0;
	int* nodeGate_h;
	int* nodeInput_h;
	int* nodeOutput_h;

	numNodePtrs = read_input_one_two_four(&nodePtrs_h, input_filename1);
	numTotalNeighbors_h = read_input_one_two_four(&nodeNeighbors_h, input_filename2);
	numNodes = read_input_three(&nodeVisited_h, &nodeGate_h, &nodeInput_h, &nodeOutput_h, input_filename3);
	numCurrLevelNodes = read_input_one_two_four(&currLevelNodes_h, input_filename4);


	//output
	int* nextLevelNodes_h = (int*)malloc(numNodePtrs * sizeof(int));

	//start clock
	clock_t begin = clock();

	// Loop over all nodes in the current level
	for (int idx = 0; idx < numCurrLevelNodes; idx++) {
		int node = currLevelNodes_h[idx];
		// Loop over all neighbors of the node
		for (int nbrIdx = nodePtrs_h[node]; nbrIdx < nodePtrs_h[node + 1]; nbrIdx++) {
			int neighbor = nodeNeighbors_h[nbrIdx];
			// If the neighbor hasn't been visited yet
			if (!nodeVisited_h[neighbor]) {
				// Mark it and add it to the queue
				nodeVisited_h[neighbor] = 1;
				nodeOutput_h[neighbor] = gate_solver(nodeGate_h[neighbor], nodeOutput_h[node], nodeInput_h[neighbor]);
				nextLevelNodes_h[numNextLevelNodes_h] = neighbor;
				++(numNextLevelNodes_h);
			}
		}
	}

	clock_t end = clock();

	float time_spent = ((double)end - begin) / CLOCKS_PER_SEC * 1000;
	printf("Execution time: %f milliseconds\n", time_spent);

	//output results to respective files

	FILE* output_file_node;
	FILE* output_file_next;
	if ((output_file_node = fopen(output_node_filename, "w")) == NULL) {
		printf("Error! opening file");
		exit(1);
	}
	if ((output_file_next = fopen(output_next_node_filename, "w")) == NULL) {
		printf("Error! opening file");
		exit(1);
	}
	//first line is the length
	fprintf(output_file_node, "%d\n", numNodePtrs-1);
	for (int loop = 0; loop < numNodePtrs-1; loop++)
		fprintf(output_file_node, "%d\n", nodeOutput_h[loop]);
	fclose(output_file_node);

	fprintf(output_file_next, "%d\n", numNextLevelNodes_h);
	for(int loop = 0; loop < numNextLevelNodes_h-1; loop++)
		fprintf(output_file_next, "%d\n", nextLevelNodes_h[loop]);

	
	fclose(output_file_next);

	return 0;
}

//int main(int argc, char* argv[]) { return process_sequential(argc, argv); }
